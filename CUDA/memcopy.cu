#include <hip/hip_runtime.h>
#include <stdint.h>

#include <chrono>
using namespace std::chrono;
using nano_double = duration<double, std::nano>;

#ifdef _WIN32
#define EXPORT_API __declspec(dllexport)
#else
#define EXPORT_API
#endif

__global__ void memcopy_kernel(uint8_t *dst, const uint8_t *src, const int n) {
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  if (ix < n) {
    dst[ix] = src[ix];
  }
}

extern "C" EXPORT_API void run_benchmark(double *times, const int nsamples,
                                         const int n) {
  hipDeviceReset();

  uint8_t *dst, *src;
  hipMalloc(&dst, n);
  hipMalloc(&src, n);

  hipStream_t stream;
  hipStreamCreate(&stream);

  int nthreads = 256;
  int nblocks = (n + nthreads - 1) / nthreads;

  for (int isample = 0; isample < nsamples; ++isample) {
    auto timer = high_resolution_clock::now();
    memcopy_kernel<<<nblocks, nthreads, 0, stream>>>(dst, src, n);
    hipStreamSynchronize(stream);
    auto elapsed = high_resolution_clock::now() - timer;
    auto time_total = duration_cast<nano_double>(elapsed).count();
    times[isample] = time_total;
  }

  hipFree(src);
  hipFree(dst);

  hipStreamDestroy(stream);
}